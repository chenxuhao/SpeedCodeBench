#include "hip/hip_runtime.h"
#include <iostream>

#define MAX_BLOCK_SZ 128

#define checkCudaErrors( call) do {										\
   hipError_t err = call;												\
   if( hipSuccess != err) {											\
       fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",	\
               __FILE__, __LINE__, hipGetErrorString( err) );			\
   exit(EXIT_FAILURE);													\
   } } while (0)

#include "scan.cu"

__global__ void gpu_radix_sort_local(unsigned int* d_out_sorted,
                                     unsigned int* d_prefix_sums,
                                     unsigned int* d_block_sums,
                                     unsigned int input_shift_width,
                                     unsigned int* d_in,
                                     unsigned int d_in_len,
                                     unsigned int max_elems_per_block);

__global__ void gpu_glbl_shuffle(unsigned int* d_out,
                                 unsigned int* d_in,
                                 unsigned int* d_scan_block_sums,
                                 unsigned int* d_prefix_sums,
                                 unsigned int input_shift_width,
                                 unsigned int d_in_len,
                                 unsigned int max_elems_per_block);

// An attempt at the gpu radix sort variant described in this paper:
// https://vgc.poly.edu/~csilva/papers/cgf.pdf
extern "C"
void radix_sort(unsigned int* h_out,
                unsigned int* const h_in,
                unsigned int d_in_len) {
  unsigned int* d_in;
  unsigned int* d_out;
  checkCudaErrors(hipMalloc(&d_in, sizeof(unsigned int) * d_in_len));
  checkCudaErrors(hipMalloc(&d_out, sizeof(unsigned int) * d_in_len));
  checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(unsigned int) * d_in_len, hipMemcpyHostToDevice));
 
  unsigned int block_sz = MAX_BLOCK_SZ;
  unsigned int max_elems_per_block = block_sz;
  unsigned int grid_sz = d_in_len / max_elems_per_block;
  // Take advantage of the fact that integer division drops the decimals
  if (d_in_len % max_elems_per_block != 0)
    grid_sz += 1;
  unsigned int* d_prefix_sums;
  unsigned int d_prefix_sums_len = d_in_len;
  checkCudaErrors(hipMalloc(&d_prefix_sums, sizeof(unsigned int) * d_prefix_sums_len));
  checkCudaErrors(hipMemset(d_prefix_sums, 0, sizeof(unsigned int) * d_prefix_sums_len));
  unsigned int* d_block_sums;
  unsigned int d_block_sums_len = 4 * grid_sz; // 4-way split
  checkCudaErrors(hipMalloc(&d_block_sums, sizeof(unsigned int) * d_block_sums_len));
  checkCudaErrors(hipMemset(d_block_sums, 0, sizeof(unsigned int) * d_block_sums_len));
  unsigned int* d_scan_block_sums;
  checkCudaErrors(hipMalloc(&d_scan_block_sums, sizeof(unsigned int) * d_block_sums_len));
  checkCudaErrors(hipMemset(d_scan_block_sums, 0, sizeof(unsigned int) * d_block_sums_len));
  // shared memory consists of 3 arrays the size of the block-wise input
  //  and 2 arrays the size of n in the current n-way split (4)
  unsigned int s_data_len = max_elems_per_block;
  unsigned int s_mask_out_len = max_elems_per_block + 1;
  unsigned int s_merged_scan_mask_out_len = max_elems_per_block;
  unsigned int s_mask_out_sums_len = 4; // 4-way split
  unsigned int s_scan_mask_out_sums_len = 4;
  unsigned int shmem_sz = (s_data_len + s_mask_out_len
                          + s_merged_scan_mask_out_len
                          + s_mask_out_sums_len
                          + s_scan_mask_out_sums_len)
                          * sizeof(unsigned int);
  // for every 2 bits from LSB to MSB:
  //  block-wise radix sort (write blocks back to global memory)
  for (unsigned int shift_width = 0; shift_width <= 30; shift_width += 2) {
    gpu_radix_sort_local<<<grid_sz, block_sz, shmem_sz>>>(d_out, 
                                                          d_prefix_sums, 
                                                          d_block_sums, 
                                                          shift_width, 
                                                          d_in, 
                                                          d_in_len, 
                                                          max_elems_per_block);
    // scan global block sum array
    sum_scan_blelloch(d_scan_block_sums, d_block_sums, d_block_sums_len);

    // scatter/shuffle block-wise sorted array to final positions
    gpu_glbl_shuffle<<<grid_sz, block_sz>>>(d_in, 
                                            d_out, 
                                            d_scan_block_sums, 
                                            d_prefix_sums, 
                                            shift_width, 
                                            d_in_len, 
                                            max_elems_per_block);
  }
  checkCudaErrors(hipMemcpy(d_out, d_in, sizeof(unsigned int) * d_in_len, hipMemcpyDeviceToDevice));
  checkCudaErrors(hipFree(d_scan_block_sums));
  checkCudaErrors(hipFree(d_block_sums));
  checkCudaErrors(hipFree(d_prefix_sums));

  checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(unsigned int) * d_in_len, hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(d_out));
  checkCudaErrors(hipFree(d_in));
}
 
__global__ void gpu_radix_sort_local(unsigned int* d_out_sorted,
                                     unsigned int* d_prefix_sums,
                                     unsigned int* d_block_sums,
                                     unsigned int input_shift_width,
                                     unsigned int* d_in,
                                     unsigned int d_in_len,
                                     unsigned int max_elems_per_block) {
  // need shared memory array for:
  // - block's share of the input data (local sort will be put here too)
  // - mask outputs
  // - scanned mask outputs
  // - merged scaned mask outputs ("local prefix sum")
  // - local sums of scanned mask outputs
  // - scanned local sums of scanned mask outputs

  // for all radix combinations:
  //  build mask output for current radix combination
  //  scan mask ouput
  //  store needed value from current prefix sum array to merged prefix sum array
  //  store total sum of mask output (obtained from scan) to global block sum array
  // calculate local sorted address from local prefix sum and scanned mask output's total sums
  // shuffle input block according to calculated local sorted addresses
  // shuffle local prefix sums according to calculated local sorted addresses
  // copy locally sorted array back to global memory
  // copy local prefix sum array back to global memory

  extern __shared__ unsigned int shmem[];
  unsigned int* s_data = shmem;
  // s_mask_out[] will be scanned in place
  unsigned int s_mask_out_len = max_elems_per_block + 1;
  unsigned int* s_mask_out = &s_data[max_elems_per_block];
  unsigned int* s_merged_scan_mask_out = &s_mask_out[s_mask_out_len];
  unsigned int* s_mask_out_sums = &s_merged_scan_mask_out[max_elems_per_block];
  unsigned int* s_scan_mask_out_sums = &s_mask_out_sums[4];

  unsigned int thid = threadIdx.x;

  // Copy block's portion of global input data to shared memory
  unsigned int cpy_idx = max_elems_per_block * blockIdx.x + thid;
  if (cpy_idx < d_in_len)
    s_data[thid] = d_in[cpy_idx];
  else
    s_data[thid] = 0;

  __syncthreads();

  // To extract the correct 2 bits, we first shift the number
  //  to the right until the correct 2 bits are in the 2 LSBs,
  //  then mask on the number with 11 (3) to remove the bits
  //  on the left
  unsigned int t_data = s_data[thid];
  unsigned int t_2bit_extract = (t_data >> input_shift_width) & 3;

  for (unsigned int i = 0; i < 4; ++i) {
    // Zero out s_mask_out
    s_mask_out[thid] = 0;
    if (thid == 0)
      s_mask_out[s_mask_out_len - 1] = 0;

    __syncthreads();

    // build bit mask output
    bool val_equals_i = false;
    if (cpy_idx < d_in_len) {
      val_equals_i = t_2bit_extract == i;
      s_mask_out[thid] = val_equals_i;
    }
    __syncthreads();

    // Scan mask outputs (Hillis-Steele)
    int partner = 0;
    unsigned int sum = 0;
    unsigned int max_steps = (unsigned int) log2f(max_elems_per_block);
    for (unsigned int d = 0; d < max_steps; d++) {
      partner = thid - (1 << d);
      if (partner >= 0) {
        sum = s_mask_out[thid] + s_mask_out[partner];
      } else {
        sum = s_mask_out[thid];
      }
      __syncthreads();
      s_mask_out[thid] = sum;
      __syncthreads();
    }

    // Shift elements to produce the same effect as exclusive scan
    unsigned int cpy_val = 0;
    cpy_val = s_mask_out[thid];
    __syncthreads();
    s_mask_out[thid + 1] = cpy_val;
    __syncthreads();
    if (thid == 0) {
      // Zero out first element to produce the same effect as exclusive scan
      s_mask_out[0] = 0;
      unsigned int total_sum = s_mask_out[s_mask_out_len - 1];
      s_mask_out_sums[i] = total_sum;
      d_block_sums[i * gridDim.x + blockIdx.x] = total_sum;
    }
    __syncthreads();
    if (val_equals_i && (cpy_idx < d_in_len)) {
      s_merged_scan_mask_out[thid] = s_mask_out[thid];
    }
    __syncthreads();
  }

  // Scan mask output sums
  // Just do a naive scan since the array is really small
  if (thid == 0) {
    unsigned int run_sum = 0;
    for (unsigned int i = 0; i < 4; ++i) {
      s_scan_mask_out_sums[i] = run_sum;
      run_sum += s_mask_out_sums[i];
    }
  }
  __syncthreads();
  if (cpy_idx < d_in_len) {
    // Calculate the new indices of the input elements for sorting
    unsigned int t_prefix_sum = s_merged_scan_mask_out[thid];
    unsigned int new_pos = t_prefix_sum + s_scan_mask_out_sums[t_2bit_extract];

    __syncthreads();

    // Shuffle the block's input elements to actually sort them
    // Do this step for greater global memory transfer coalescing
    //  in next step
    s_data[new_pos] = t_data;
    s_merged_scan_mask_out[new_pos] = t_prefix_sum;

    __syncthreads();

    // Copy block - wise prefix sum results to global memory
    // Copy block-wise sort results to global 
    d_prefix_sums[cpy_idx] = s_merged_scan_mask_out[thid];
    d_out_sorted[cpy_idx] = s_data[thid];
  }
}
 
__global__ void gpu_glbl_shuffle(unsigned int* d_out,
                                 unsigned int* d_in,
                                 unsigned int* d_scan_block_sums,
                                 unsigned int* d_prefix_sums,
                                 unsigned int input_shift_width,
                                 unsigned int d_in_len,
                                 unsigned int max_elems_per_block) {
  // get d = digit
  // get n = blockIdx
  // get m = local prefix sum array value
  // calculate global position = P_d[n] + m
  // copy input element to final position in d_out

  unsigned int thid = threadIdx.x;
  unsigned int cpy_idx = max_elems_per_block * blockIdx.x + thid;

  if (cpy_idx < d_in_len) {
    unsigned int t_data = d_in[cpy_idx];
    unsigned int t_2bit_extract = (t_data >> input_shift_width) & 3;
    unsigned int t_prefix_sum = d_prefix_sums[cpy_idx];
    unsigned int data_glbl_pos = d_scan_block_sums[t_2bit_extract * gridDim.x + blockIdx.x]
      + t_prefix_sum;
    __syncthreads();
    d_out[data_glbl_pos] = t_data;
  }
}

