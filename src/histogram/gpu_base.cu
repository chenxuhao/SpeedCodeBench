#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "util.h"

__global__ void histo_prescan_kernel (
        unsigned int* input,
        int size,
        unsigned int* minmax);

__global__ void histo_main_kernel (
        uchar4 *sm_mappings,
        unsigned int num_elements,
        unsigned int sm_range_min,
        unsigned int sm_range_max,
        unsigned int histo_height,
        unsigned int histo_width,
        unsigned int *global_subhisto,
        unsigned int *global_histo,
        unsigned int *global_overflow);

__global__ void histo_intermediates_kernel (
        uint2 *input,
        unsigned int height,
        unsigned int width,
        unsigned int input_pitch,
        uchar4 *sm_mappings);

__global__ void histo_final_kernel (
        unsigned int sm_range_min,
        unsigned int sm_range_max,
        unsigned int histo_height,
        unsigned int histo_width,
        unsigned int *global_subhisto,
        unsigned int *global_histo,
        unsigned int *global_overflow,
        unsigned int *final_histo);

/******************************************************************************
* Implementation: GPU
* Details:
* in the GPU implementation of histogram, we begin by computing the span of the
* input values into the histogram. Then the histogramming computation is carried
* out by a (BLOCK_X, BLOCK_Y) sized grid, where every group of Y (same X)
* computes its own partial histogram for a part of the input, and every Y in the
* group exclusively writes to a portion of the span computed in the beginning.
* Finally, a reduction is performed to combine all the partial histograms into
* the final result.
******************************************************************************/
extern "C" 
void histogram(int n,
               unsigned int img_width, unsigned int img_height,
               unsigned int* image,
               unsigned int histo_width, unsigned int histo_height,
               unsigned char* histo) {
  char *prescans = "PreScanKernel";
  char *postpremems = "PostPreMems";
  char *intermediates = "IntermediatesKernel";
  char *mains = "MainKernel";
  char *finals = "FinalKernel";
  
  int even_width = ((img_width+1)/2)*2;
  unsigned int* input;
  unsigned int* ranges;
  uchar4* sm_mappings;
  unsigned int* global_subhisto;
  unsigned short* global_histo;
  unsigned int* global_overflow;
  unsigned char* final_histo;

  hipMalloc((void**)&input           , even_width*(((img_height+UNROLL-1)/UNROLL)*UNROLL)*sizeof(unsigned int));
  hipMalloc((void**)&ranges          , 2*sizeof(unsigned int));
  hipMalloc((void**)&sm_mappings     , img_width*img_height*sizeof(uchar4));
  hipMalloc((void**)&global_subhisto , img_width*histo_height*sizeof(unsigned int));
  hipMalloc((void**)&global_histo    , img_width*histo_height*sizeof(unsigned short));
  hipMalloc((void**)&global_overflow , img_width*histo_height*sizeof(unsigned int));
  hipMalloc((void**)&final_histo     , img_width*histo_height*sizeof(unsigned char));
  hipMemset(final_histo           ,0 , img_width*histo_height*sizeof(unsigned char));
  for (int y=0; y < img_height; y++){
    hipMemcpy(&(((unsigned int*)input)[y*even_width]),&image[y*img_width],img_width*sizeof(unsigned int), hipMemcpyHostToDevice);
  }
  unsigned int *zeroData = (unsigned int *) calloc(img_width*histo_height, sizeof(unsigned int));
  for (int iter = 0; iter < n; iter++) {
    unsigned int ranges_h[2] = {UINT32_MAX, 0};
    hipMemcpy(ranges,ranges_h, 2*sizeof(unsigned int), hipMemcpyHostToDevice);
    histo_prescan_kernel<<<dim3(PRESCAN_BLOCKS_X),dim3(PRESCAN_THREADS)>>>((unsigned int*)input, img_height*img_width, ranges);
    hipMemcpy(ranges_h,ranges, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(global_subhisto,zeroData, img_width*histo_height*sizeof(unsigned int), hipMemcpyHostToDevice);
    histo_intermediates_kernel<<<dim3((img_height + UNROLL-1)/UNROLL), dim3((img_width+1)/2)>>>(
                (uint2*)(input),
                (unsigned int)img_height,
                (unsigned int)img_width,
                (img_width+1)/2,
                (uchar4*)(sm_mappings)
    );
    histo_main_kernel<<<dim3(BLOCK_X, ranges_h[1]-ranges_h[0]+1), dim3(THREADS)>>>(
                (uchar4*)(sm_mappings),
                img_height*img_width,
                ranges_h[0], ranges_h[1],
                histo_height, histo_width,
                (unsigned int*)(global_subhisto),
                (unsigned int*)(global_histo),
                (unsigned int*)(global_overflow)
    );
    histo_final_kernel<<<dim3(BLOCK_X*3), dim3(512)>>>(
                ranges_h[0], ranges_h[1],
                histo_height, histo_width,
                (unsigned int*)(global_subhisto),
                (unsigned int*)(global_histo),
                (unsigned int*)(global_overflow),
                (unsigned int*)(final_histo)
    );
  }
  hipMemcpy(histo,final_histo, histo_height*histo_width*sizeof(unsigned char), hipMemcpyDeviceToHost);
  hipFree(input);
  hipFree(ranges);
  hipFree(sm_mappings);
  hipFree(global_subhisto);
  hipFree(global_histo);
  hipFree(global_overflow);
  hipFree(final_histo);
}
