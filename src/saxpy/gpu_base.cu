
#include <hip/hip_runtime.h>

__global__ void saxpy_kernel(const int n, const float a, const float *x, float *y) {
  // Get our global thread ID
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // Make sure we do not go out of bounds
  if (i < n)
    y[i] = a * x[i] + y[i];
}

extern "C"
void saxpy(const int n, const float a, const float *x, float *y) {
  // Size, in bytes, of each vector
  int64_t bytes = int64_t(n) * sizeof(float);

  // Allocate memory for each vector on GPU
  float *d_x, *d_y;
  hipMalloc(&d_x, bytes);
  hipMalloc(&d_y, bytes);
  hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);

  int blockSize, gridSize;

  // Number of threads in each thread block
  blockSize = 1024;

  // Number of thread blocks in grid
  gridSize = (int)ceil((float)n/blockSize);

  // Execute the kernel
  saxpy_kernel<<<gridSize, blockSize>>>(n, a, d_x, d_y);

  // Copy array back to host
  hipMemcpy( y, d_y, bytes, hipMemcpyDeviceToHost );

  // Release device memory
  hipFree(d_x);
  hipFree(d_y);
}
