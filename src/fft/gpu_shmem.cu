/*
Author: Sara Baghsorkhi.
This implementation is partly based on the SC08 paper by Naga K. Govindaraju et al.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  return -1; }}

// Block index
#define  bx  blockIdx.x
#define  by  blockIdx.y
// Thread index
#define tx  threadIdx.x

#define DEBUG 0
#define B 1024 
#define EMUL 0

#define R2 1
#define R4 0
#define R8 0
#define R16 0

#if R2
#define N 4*4*4*4
#define R 2
#endif

#if R4
#define N 4*4*4*4
#define R 4
#endif

#if R8
#define N
#define R 8
#endif

#if R16
#define N 4*4*4*4
#define R 16
#endif

#define T  N/R 

inline __host__ __device__ float2 operator*( float2 a, float2 b ) { return make_float2( a.x*b.x-a.y*b.y, a.x*b.y+a.y*b.x ); }
inline __host__ __device__ float2 operator+( float2 a, float2 b ) { return make_float2( a.x + b.x, a.y + b.y ); }
inline __host__ __device__ float2 operator-( float2 a, float2 b ) { return make_float2( a.x - b.x, a.y - b.y ); }
inline __host__ __device__ float2 operator*( float2 a, float b )  { return make_float2( b*a.x , b*a.y); }

#define COS_PI_8  0.923879533f
#define SIN_PI_8  0.382683432f
#define exp_1_16  make_float2(  COS_PI_8, -SIN_PI_8 )
#define exp_3_16  make_float2(  SIN_PI_8, -COS_PI_8 )
#define exp_5_16  make_float2( -SIN_PI_8, -COS_PI_8 )
#define exp_7_16  make_float2( -COS_PI_8, -SIN_PI_8 )
#define exp_9_16  make_float2( -COS_PI_8,  SIN_PI_8 )
#define exp_1_8   make_float2(  1, -1 )//requires post-multiply by 1/sqrt(2)
#define exp_1_4   make_float2(  0, -1 )
#define exp_3_8   make_float2( -1, -1 )//requires post-multiply by 1/sqrt(2)
  
void FFT2( float2* v ) { 
  float2 v0 = v[0];  
  v[0] = v0 + v[1]; 
  v[1] = v0 - v[1]; 
}

__device__ void GPU_FFT2( float2 &v1,float2 &v2 ) { 
  float2 v0 = v1;  
  v1 = v0 + v2; 
  v2 = v0 - v2; 
}

__device__ void GPU_FFT4( float2 &v0,float2 &v1,float2 &v2,float2 &v3) { 
  GPU_FFT2(v0, v2);
  GPU_FFT2(v1, v3);
  v3 = v3 * exp_1_4;
  GPU_FFT2(v0, v1);
  GPU_FFT2(v2, v3);    
}

inline __device__ void GPU_FFT2(float2* v) {
  GPU_FFT2(v[0],v[1]);
}

inline __device__ void GPU_FFT4(float2* v) {
  GPU_FFT4(v[0],v[1],v[2],v[3] );
}

inline __device__ void GPU_FFT8(float2* v) {
  GPU_FFT2(v[0],v[4]);
  GPU_FFT2(v[1],v[5]);
  GPU_FFT2(v[2],v[6]);
  GPU_FFT2(v[3],v[7]);

  v[5]=(v[5]*exp_1_8)*M_SQRT1_2;
  v[6]=v[6]*exp_1_4;
  v[7]=(v[7]*exp_3_8)*M_SQRT1_2;

  GPU_FFT4(v[0],v[1],v[2],v[3]);
  GPU_FFT4(v[4],v[5],v[6],v[7]);
}

inline __device__ void GPU_FFT16( float2 *v ) {
    GPU_FFT4( v[0], v[4], v[8], v[12] );
    GPU_FFT4( v[1], v[5], v[9], v[13] );
    GPU_FFT4( v[2], v[6], v[10], v[14] );
    GPU_FFT4( v[3], v[7], v[11], v[15] );

    v[5]  = (v[5]  * exp_1_8 ) * M_SQRT1_2;
    v[6]  =  v[6]  * exp_1_4;
    v[7]  = (v[7]  * exp_3_8 ) * M_SQRT1_2;
    v[9]  =  v[9]  * exp_1_16;
    v[10] = (v[10] * exp_1_8 ) * M_SQRT1_2;
    v[11] =  v[11] * exp_3_16;
    v[13] =  v[13] * exp_3_16;
    v[14] = (v[14] * exp_3_8 ) * M_SQRT1_2;
    v[15] =  v[15] * exp_9_16;

    GPU_FFT4( v[0],  v[1],  v[2],  v[3] );
    GPU_FFT4( v[4],  v[5],  v[6],  v[7] );
    GPU_FFT4( v[8],  v[9],  v[10], v[11] );
    GPU_FFT4( v[12], v[13], v[14], v[15] );
}

__device__ int GPU_expand(int idxL, int N1, int N2 ) { 
  return (idxL/N1)*N1*N2 + (idxL%N1); 
}      

__device__ void GPU_exchange( float2* v, int stride, int idxD, int incD, int idxS, int incS ) {
  __shared__ float work[T*R*2];//T*R*2
  float* sr = work;
  float* si = work+T*R;  
  __syncthreads(); 
  for( int r=0; r<R; r++ ) { 
    int i = (idxD + r*incD)*stride; 
    sr[i] = v[r].x;
    si[i] = v[r].y;  
  }   
  __syncthreads(); 

  for( int r=0; r<R; r++ ) { 
    int i = (idxS + r*incS)*stride;     
    v[r] = make_float2(sr[i], si[i]);  
  }        
}      

  
__device__ void GPU_DoFft(float2* v, int j, int stride=1) { 
  for ( int Ns=1; Ns<N; Ns*=R ) {
    float angle = -2*M_PI*(j%Ns)/(Ns*R); 
    for ( int r=0; r<R; r++ ) {
      v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
    }
#if R2
    GPU_FFT2( v );
#endif

#if R4
    GPU_FFT4( v );
#endif

#if R8
    GPU_FFT8( v );	
#endif

#if R16
    GPU_FFT16( v );
#endif
    int idxD = GPU_expand(j,Ns,R); 
    int idxS = GPU_expand(j,N/R,R); 
    GPU_exchange( v,stride, idxD,Ns, idxS,N/R );
  }      
}    

__global__ void GPU_FftShMem(float2* data) { 
  float2 v[R];
  data+=bx*N; 
  int idxG = tx; 
  for ( int r=0; r<R; r++ ) {  
    v[r] = data[idxG + r*T];
  } 
  GPU_DoFft( v, tx );  
  for ( int r=0; r<R; r++ )
    data[idxG + r*T] = v[r]; 
}

void fft(float2 *dst, float2 *source, int batch, int n) {
  int nthreads = T;
  // allocate device memory
  int64_t n_bytes = n*batch*sizeof(float2);
  float *shared_source =(float *)malloc(n_bytes);  
  float2 *d_source, *d_work;
  float *d_shared_source;
  hipMalloc((void**) &d_shared_source, n_bytes);
  // copy host memory to device
  hipMemcpy(d_shared_source, shared_source, n_bytes,hipMemcpyHostToDevice);
  hipMalloc((void**) &d_source, n_bytes);
  // copy host memory to device
  hipMemcpy(d_source, source, n_bytes,hipMemcpyHostToDevice);
  hipMalloc((void**) &d_work, n_bytes);
  hipMemset(d_work, 0,n_bytes);
  GPU_FftShMem<<<dim3(batch), dim3(nthreads)>>>(d_source); 	
  // copy device memory to host
  hipMemcpy(dst, d_source, n_bytes,hipMemcpyDeviceToHost);
  hipFree(d_source);
  hipFree(d_work);
  free(shared_source);  
}

