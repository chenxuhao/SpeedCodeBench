#include "hip/hip_runtime.h"
// Copyright 2022 MIT
// Authors: Xuhao Chen <cxh@mit.edu>
#include "common_kernels.cuh"
#include "cuda_launch_config.hpp"

__global__ void compute_error(GraphGPU g, latent_t *latents, 
                              score_t lambda, score_t step, 
                              score_t *errors, score_t *squared_errors) {
  __shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];

  const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
  const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
  const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
  const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
  const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

  for (int u = warp_id; u < g.V(); u += num_warps) {
    latent_t *u_latent = &latents[K*u];
    score_t *u_err = &errors[K*u];
    if (thread_lane < 2)
      ptrs[warp_lane][thread_lane] = g.edge_begin(u+thread_lane);
    auto row_begin = ptrs[warp_lane][0];
    auto row_end   = ptrs[warp_lane][1];
    for (auto offset = row_begin; offset < row_end; offset ++) {
      auto v = g.getEdgeDst(offset);
      auto v_latent = &latents[v*K];
      latent_t temp_p[(K-1)/WARP_SIZE + 1];
      latent_t temp_q[(K-1)/WARP_SIZE + 1];
      score_t estimate = 0;
      for (int i = 0; i < K; i += WARP_SIZE) {
        if (thread_lane+i < K) {
          int j = i/WARP_SIZE;
          temp_p[j] = u_latent[thread_lane+i];
          temp_q[j] = v_latent[thread_lane+i];
          estimate += temp_p[j] * temp_q[j];
        }
      }
      warp_reduce(estimate);
      /*
      estimate += SHFL_DOWN(estimate, 16);
      estimate += SHFL_DOWN(estimate, 8);
      estimate += SHFL_DOWN(estimate, 4);
      estimate += SHFL_DOWN(estimate, 2);
      estimate += SHFL_DOWN(estimate, 1);
      estimate = SHFL(estimate, 0);
      //*/
      score_t delta = g.getEdgeData(offset) - estimate;
      if (thread_lane == 0) squared_errors[u] += delta * delta;
      for (int i = 0; i < K; i += WARP_SIZE) {
        if (thread_lane+i < K) {
          int j = i/WARP_SIZE;
          u_err[thread_lane+i] += temp_q[j] * delta;
        }
      }
    }
  }
}

void SGDSolver(BipartiteGraph &g, std::vector<latent_t> &latents, int *h_ordering) {
  size_t memsize = print_device_info(0);
  auto nv = g.V();
  auto ne = g.E();
  //auto num_users = g.V(0);
  //auto num_items = g.V(1);
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";

  GraphGPU gg(g);
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = (nv-1) / WARPS_PER_BLOCK + 1;
  if (nblocks > 65536) nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(compute_error, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  //size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  //nblocks = std::min(max_blocks, nblocks);
  std::cout << "CUDA CF (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
  //for (size_t i = 0; i < latents.size(); i++) std::cout << latents[i] << "\n";

  latent_t *h_latents = &latents[0];
  latent_t *d_latents;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_latents, nv * K * sizeof(latent_t)));
  CUDA_SAFE_CALL(hipMemcpy(d_latents, h_latents, nv * K * sizeof(latent_t), hipMemcpyHostToDevice));
  score_t h_total_error = 0, *d_total_error, *d_errors, *squared_errors;
  CUDA_SAFE_CALL(hipMalloc((void **)&squared_errors, nv * sizeof(score_t)));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total_error, sizeof(score_t)));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_errors, nv * K * sizeof(score_t)));
  init_const_float<<<(nv*K-1)/nthreads+1, nthreads>>>(nv*K, 0.0, d_errors);
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  Timer t;
  t.Start();
  int iter = 0;
  do {
    ++iter;
    h_total_error = 0.0;
    init_const_float<<<(nv-1)/nthreads+1, nthreads>>>(nv, 0.0, squared_errors);
    CUDA_SAFE_CALL(hipMemcpy(d_total_error, &h_total_error, sizeof(score_t), hipMemcpyHostToDevice));
    compute_error<<<nblocks, nthreads>>>(gg, d_latents, lambda, step, d_errors, squared_errors);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    update_vertex<<<(nv-1)/nthreads+1, nthreads>>>(gg, lambda, step, d_latents, d_errors);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    rmse<<<(nv-1)/nthreads+1, nthreads>>>(nv, squared_errors, d_total_error);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(&h_total_error, d_total_error, sizeof(score_t), hipMemcpyDeviceToHost));
    printf("iteration %d: RMSE error = %f\n", iter, sqrt(h_total_error/ne));
    //CUDA_SAFE_CALL(hipMemcpy(h_latents, d_latents, nv*K*sizeof(latent_t), hipMemcpyDeviceToHost));
  } while (iter < max_iters && h_total_error > cf_epsilon);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "iterations = " << iter << ".\n";
  std::cout << "runtime [gpu_base] = " << t.Seconds() << " sec\n";

  CUDA_SAFE_CALL(hipMemcpy(h_latents, d_latents, nv*K*sizeof(latent_t), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(d_latents));
  CUDA_SAFE_CALL(hipFree(d_total_error));
  CUDA_SAFE_CALL(hipFree(d_errors));
  CUDA_SAFE_CALL(hipFree(squared_errors));
}

