#include "hip/hip_runtime.h"
#include "ctimer.h"
#include <stdint.h>
typedef float DType;
typedef int64_t vidType;
typedef uint32_t eidType;


// SPMM with CSR.
// C = SpMM(SpA, B) + C
// BT is transposed B
__global__ void SpMMCsrKernel(vidType m, eidType nnz, int n,
                              const eidType* __restrict__ Ap,
                              const vidType* __restrict__ Aj,
                              const DType* __restrict__ Ax,
                              const DType* __restrict__ BT,
                              DType* __restrict__ C) {
  int tx = blockIdx.y * blockDim.x + threadIdx.x;
  int ty = blockIdx.x * blockDim.y + threadIdx.y;
  const int stride_y = blockDim.y * gridDim.x;
  const int stride_x = blockDim.x * gridDim.y;
  while (ty < m) {
    while (tx < n) {
      DType sum = 0;
      for (int off = Ap[ty]; off < Ap[ty+1]; off++) {
        vidType k = Aj[off]; // column id
        DType value = Ax[off];   // A[i][k]
        //sum += value * B[k*n + tx]; // A[i][k] * B[k][j]
        sum += value * BT[tx*m + k]; // A[i][k] * BT[j][k]
      }
      C[ty * n + tx] += sum;
      tx += stride_x;
    }
    ty += stride_y;
  }
}

// A: m x m 
// B: m x n
// BT:n x m
// C: m x n
extern "C"
void SpmDm(char transa, char transb, 
           vidType m, eidType nnz, int n,
           DType alpha, const eidType *Ap,
           const vidType *Aj, const DType *Ax, 
           int lda, const DType *BT, int ldb, 
           DType beta, DType *C, int ldc) {
  printf("CUDA SpMDM solver\n");
  ctimer_t t;
  ctimer_start(&t);
  eidType * d_Ap;
  vidType * d_Aj;
  DType *d_Ax, *d_BT, *d_C;

  hipMalloc(&d_Ap, sizeof(eidType)*(m+1));
  hipMalloc(&d_Aj, sizeof(vidType)*nnz);
  hipMalloc(&d_Ax, sizeof(DType)*nnz);
  hipMalloc(&d_BT, sizeof(DType)*m*n);
  hipMalloc(&d_C, sizeof(DType)*m*n);
  hipMemcpy(d_Ap, Ap, sizeof(eidType)*(m+1), hipMemcpyHostToDevice);
  hipMemcpy(d_Aj, Aj, sizeof(vidType)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(d_Ax, Ax, sizeof(DType)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(d_BT, BT, sizeof(DType)*m*n, hipMemcpyHostToDevice);
  dim3 blockSize(32, 32);
  int nrows = (int)ceil((float)m/32);
  int ncols = (int)ceil((float)n/32);
  dim3 gridSize(nrows, ncols);

  SpMMCsrKernel<<<gridSize, blockSize>>>(m, nnz, n, d_Ap, d_Aj, d_Ax, d_BT, d_C);

  ctimer_stop(&t);
  ctimer_measure(&t);
  ctimer_print(t, "SpmDm");
  //float gbyte = bytes_per_spmdm(m, nnz) / 10e9;
  //float GFLOPs = 2*nnz / time / 10e9;
  //float GBYTEs = gbyte / time;
  //printf("Throughput: compute %5.2f GFLOP/s, memory %5.1f GB/s\n", GFLOPs, GBYTEs);
  hipMemcpy(C, d_C, m*n*sizeof(DType), hipMemcpyDeviceToHost);
  hipFree(d_Ap);
  hipFree(d_Aj);
  hipFree(d_Ax);
  hipFree(d_BT);
  hipFree(d_C);
}

